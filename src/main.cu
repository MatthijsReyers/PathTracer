#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>

#include "vec3.h"
// #include "ray.h"
// #include "sphere.h"
// #include "hittable.h"
// #include "hitable_list.h"
// #include "camera.h"
// #include "image.h"

#define WIDTH 600
#define HEIGHT 400
#define SAMPLES 50
#define CAMERA_DISTANCE -1
#define RENDERPLANE_SCALE 4

__global__
void renderPixel(int n, vec3 *pixels)
{
	pixels[n] = vec3(1.0, 0, 0);
}

int main(int argv, char** argc)
{
	vec3 *pixels;
	hipMallocManaged(&pixels, WIDTH*HEIGHT*sizeof(vec3));

	renderPixel<<<1, 1>>>( int(WIDTH*HEIGHT), pixels);

	hipDeviceSynchronize();

	std::ofstream file;
	file.open("out.ppm");
	file << "P3\n" << WIDTH << " " << HEIGHT << "\n255\n";

    for (int i = WIDTH*HEIGHT; i > 0; i--)
    {
        vec3 pixel = pixels[i];

        int red = int(255.99 * pixel.r());
        int green = int(255.99 * pixel.g());
        int blue = int(255.99 * pixel.b());

        file << red << " " << green << " " << blue << "\n";
    }

    file.close();
}

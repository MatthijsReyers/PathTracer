
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <chrono>

// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	printf("- ");
	y[n] = std::sqrt(x[n]);
}

int main(void)
{
	hipError_t error  = hipGetLastError();
	if (error != 0) {
		std::cout << "ERROR: could not start program CUDA gave error: " << hipGetErrorString(error) << std::endl;
		return 1;
	}

	int N = 1 << 16;
	float *x, *y;

	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 908397534.0f;
		y[i] = 0.0f;
	}

	add<<<5, 256>>>(N, x, y);

	hipDeviceSynchronize();

	hipFree(x);
	hipFree(y);
}

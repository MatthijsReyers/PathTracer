
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <math.h>
#include <unistd.h>

#define WIDTH 600
#define HEIGHT 400

__global__
void renderPixel(int n, float *pixels)
{
	for (int i = 0; i < n; i++)
		for (int a = 0; a < 666; a++)
			std::sqrt(n);

	// pixels[n*3+0] = 1.0f;
	// pixels[n*3+1] = 0.0f;
	// pixels[n*3+2] = 1.0f;
}

int main(int argv, char** argc)
{
	float *pixels;

	std::cout << "Cuda error state: " << hipGetLastError() << std::endl;
	std::cout << "Cuda error state: " << hipGetLastError() << std::endl;

	hipMallocManaged(&pixels, 3*WIDTH*HEIGHT*sizeof(float));

	std::cout << "Starting this many cuda cores: " << WIDTH*HEIGHT << std::endl;

	renderPixel<<<1, 1>>>(WIDTH*HEIGHT, pixels);

	std::cout << "Waiting for devices to sync..." << std::endl;

	usleep(100000);

	hipDeviceSynchronize();

	std::cout << "Done!" << std::endl;

	std::ofstream file;
	file.open("out.ppm");
	file << "P3\n" << WIDTH << " " << HEIGHT << "\n255\n";

    for (int i = WIDTH*HEIGHT; i > 0; i--)
    {
        int red = int(255.99 * pixels[i*3+0]);
        int green = int(255.99 * pixels[i*3+1]);
        int blue = int(255.99 * pixels[i*3+2]);

        file << red << " " << green << " " << blue << "\n";
        // std::cout << red << " " << green << " " << blue << "\n";
    }

	hipFree(pixels);

    file.close();
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>

#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hittable.h"
#include "hitable_list.h"
#include "camera.h"
#include "image.h"

#define WIDTH 600
#define HEIGHT 400
#define SAMPLES 50
#define CAMERA_DISTANCE -1
#define RENDERPLANE_SCALE 4


__host__ __device__
inline vec3 color(const ray& r, hittable *world)
{
    hit_record rec;

	if (world->hit(r, 0.0, MAXFLOAT, rec))
        return 0.5*vec3(rec.normal.x()+1, rec.normal.y()+1, rec.normal.z()+1);

	else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5*(unit_direction.y() + 1.0);
        return (1.0-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
}

__global__
void renderPixel(int n, hittable_list *world, camera cam, image output)
{
	int w = (n % WIDTH);
	int h = n / WIDTH;

	vec3 col(1, 0, 1);
	// for (int s = 0; s < SAMPLES; s++)
	// {
	// 	float u = float(w) + (float(s/SAMPLES) / float(WIDTH));
	// 	float v = float(h) + (float(s/SAMPLES) / float(HEIGHT));
	// 	ray r = cam.get_ray(u, v);
	// 	col += color(r, world);
	// }
	// col /= float(SAMPLES);

	output.setPixel(w, h, col.e[0], col.e[1], col.e[2]);
}

int main(int argv, char** argc)
{
	vec3 *pixels;
	hipMallocManaged(&pixels, w*h*sizeof(vec3));

	renderPixel<<<1, 1>>>( int(WIDTH*HEIGHT), pixels);

	hipDeviceSynchronize();

	// hittable *list[2];
    // list[0] = new sphere(vec3(0,0,-1), 0.5);
    // list[1] = new sphere(vec3(0,-100.5,-1), 100);
    // hittable_list *world = new hittable_list(list,2);

	// image output(WIDTH, HEIGHT);
	// camera cam(WIDTH, HEIGHT, CAMERA_DISTANCE, RENDERPLANE_SCALE);

	// for (int h = 0; h < HEIGHT; h++)
	// {
	// 	for (int w = 0; w < WIDTH; w++)
	// 	{
	// 		vec3 col(0, 0, 0);
	// 		for (int s = 0; s < SAMPLES; s++)
	// 		{
    //             float u = float(w + (rand() / (RAND_MAX + 1.0))) / float(WIDTH);
    //             float v = float(h + (rand() / (RAND_MAX + 1.0))) / float(HEIGHT);
    //             ray r = cam.get_ray(u, v);
    //             col += color(r, world);
    //         }
    //         col /= float(SAMPLES);
			
	// 		output.setPixel(w, h, col.r(), col.g(), col.b());
	// 	}
	// }

	// output.write("out.ppm");
	// output.close();
}

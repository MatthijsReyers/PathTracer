
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <chrono>

// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	for (int i = 0; i < n; i++)
		for (int a = 0; a < 666; a++)
			y[i] = std::sqrt(x[i]);
}

void time()
{
	int now = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    std::cout << now << std::endl;
}

int main(void)
{
	hipError_t error  = hipGetLastError();
	if (error != 0) {
		std::cout << "ERROR: could not start program CUDA gave error: " << hipGetErrorString(error) << std::endl;
		return 1;
	}

	int N = 1 << 16;
	float *x, *y;

	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 908397534.0f;
		y[i] = 0.0f;
	}

	time();

	add<<<1, 1>>>(N, x, y);

	hipDeviceSynchronize();

	for (int i = 0; i < N; i++)
		std::cout << y[i] << std::endl;

	time();

	hipFree(x);
	hipFree(y);
}

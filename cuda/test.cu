
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <chrono>

#define DIMENSION 512

// CUDA kernel to add elements of two arrays
__global__
void render(int n, float *x, float *y, float *pixels)
{
	x[n] = 500.0;
	// pixels[n*3+0] = 0.0;
	// pixels[n*3+1] = 0.5;
	// pixels[n*3+2] = 1.0;
}

void time()
{
	int now = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    std::cout << now << std::endl;
}

int main(void)
{
	hipError_t error  = hipGetLastError();
	if (error != 0) {
		std::cout << "ERROR: could not start program CUDA gave error: " << hipGetErrorString(error) << std::endl;
		return 1;
	}

	int N = 1 << 18;
	float *x, *y, *pixels;

	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));
	hipMallocManaged(&pixels, 3*N*sizeof(float));

	error  = hipGetLastError();
	if (error != 0) {
		std::cout << "ERROR: this happened: " << hipGetErrorString(error) << std::endl;
		return 1;
	}

	for (int i = 0; i < N; i++) {
		x[i] = 2.0f;
		y[i] = 1.0f;
		pixels[i] = 0.0f;
	}

	time();

	render<<<1, 1>>>(N, x, y, pixels);

	hipDeviceSynchronize();

	time();

	error  = hipGetLastError();
	if (error != 0) {
		std::cout << "ERROR: this happened: " << hipGetErrorString(error) << std::endl;
		return 1;
	}

	for (int i = 0; i < N; i++) {
		std::cout << x[i] << " " << y[i] << std::endl;
	}

	// std::ofstream file;
	// file.open("out.ppm");
	// file << "P3\n" << DIMENSION << " " << DIMENSION << "\n255\n";
	// std::cout << "writing file\n";
    // for (int i = DIMENSION*DIMENSION -1; i > 0; i--)
    // {
    //     int red = int(255.99 * pixels[i*3+0]);
    //     int green = int(255.99 * pixels[i*3+1]);
    //     int blue = int(255.99 * pixels[i*3+2]);

    //     file << red << " " << green << " " << blue << "\n";
    // }
	// file.close();
	
	hipFree(x);
	hipFree(y);
	hipFree(pixels);
}

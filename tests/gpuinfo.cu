
int main(int argc, char const *argv[])
{
    int count;
    hipDeviceProp_t props;

    hipGetDeviceCount(&count);

    for (int i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&props, i);
        std::cout << "Device name:           " << props.name << std::endl;
        std::cout << "Clock rate:            " << props.clockRate << std::endl;
        std::cout << "Max Threads Per Block: " << props.maxThreadsPerBlock << std::endl;
        std::cout << "Max Threads Dim:       " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << std::endl;
        std::cout << "Max Grid Size:         " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << std::endl;
    }
}
